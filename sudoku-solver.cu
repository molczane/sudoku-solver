// #include <cuda_runtime.h>
// #include <stdio.h>
// #include <stdbool.h>

// #define GRID_SIZE 81
// #define THREADS_PER_BLOCK 1

// // Helper function to check if placing a number is valid
// __device__ bool is_valid(int *board, int row, int col, int num) {
//     // Check the row
//     for (int i = 0; i < 9; i++) {
//         if (board[row * 9 + i] == num) {
//             return false;
//         }
//     }

//     // Check the column
//     for (int i = 0; i < 9; i++) {
//         if (board[i * 9 + col] == num) {
//             return false;
//         }
//     }

//     // Check the 3x3 sub-grid
//     int box_row_start = (row / 3) * 3;
//     int box_col_start = (col / 3) * 3;
//     for (int i = box_row_start; i < box_row_start + 3; i++) {
//         for (int j = box_col_start; j < box_col_start + 3; j++) {
//             if (board[i * 9 + j] == num) {
//                 return false;
//             }
//         }
//     }

//     return true;
// }

// __device__ int popcount9(unsigned int x) {
//     return __popc(x); // CUDA intrinsic to count set bits
// }

// __device__ bool find_empty(int *board, int *row, int *col) {
//     // Compute row/col/cell masks from the current board
//     unsigned int row_used_numbers[9], col_used_numbers[9], cell_used_numbers[9];
//     for (int r = 0; r < 9; r++) {
//         row_used_numbers[r] = 0;
//         col_used_numbers[r] = 0;
//         cell_used_numbers[r] = 0;
//     }

//     // Set bits for existing numbers
//     for (int r = 0; r < 9; r++) {
//         for (int c = 0; c < 9; c++) {
//             int val = board[r * 9 + c];
//             if (val > 0) {
//                 unsigned int bit = 1 << (val - 1);
//                 // No validation check here; assume board is at least partially valid
//                 row_used_numbers[r] |= bit;
//                 col_used_numbers[c] |= bit;
//                 int box_idx = (r/3)*3 + (c/3);
//                 cell_used_numbers[box_idx] |= bit;
//             }
//         }
//     }

//     int min_domain_size = 10; 
//     int best_row = -1, best_col = -1;

//     for (int i = 0; i < 9; i++) {
//         for (int j = 0; j < 9; j++) {
//             int val = board[i * 9 + j];
//             if (val == 0) {
//                 // Compute used mask for this cell
//                 unsigned int used = row_used_numbers[i] | col_used_numbers[j] | cell_used_numbers[(i/3)*3+(j/3)];

//                 // possible values bitmask (1 means digit allowed)
//                 unsigned int possible = (~used) & 0x1FF;
//                 int domain_size = popcount9(possible);

//                 if (domain_size == 0) {
//                     // No possibilities for this cell, treat as domain_size=10 so we never choose this cell
//                     domain_size = 10;
//                 }

//                 // Update minimum domain
//                 if (domain_size < min_domain_size) {
//                     min_domain_size = domain_size;
//                     best_row = i;
//                     best_col = j;
//                     if (domain_size == 1) {
//                         // Perfect MRV cell, return immediately
//                         *row = best_row;
//                         *col = best_col;
//                         return true;
//                     }
//                 }
//             }
//         }
//     }

//     // If no empty cells found, puzzle is solved
//     if (best_row == -1 && best_col == -1) {
//         return false;
//     }

//     // Return the cell with minimal domain size found
//     *row = best_row;
//     *col = best_col;
//     return true;
// }


// // Device function to find the next empty cell
// // __device__ bool find_empty(int *board, int *row, int *col) {
// //     int min_domain_size = 10; // Start with a value greater than the max possible domain size (9)
// //     int best_row = -1;
// //     int best_col = -1;

// //     // Iterate over all cells
// //     for (int i = 0; i < 9; i++) {
// //         for (int j = 0; j < 9; j++) {
// //             int val = board[i * 9 + j];
// //             if (val == 0) {
// //                 // Empty cell, count possibilities
// //                 int domain_size = 0;
// //                 for (int num = 1; num <= 9; num++) {
// //                     // Check if placing 'num' is valid
// //                     bool can_place = true;
// //                     // Row check
// //                     for (int x = 0; x < 9; x++) {
// //                         if (board[i * 9 + x] == num) {
// //                             can_place = false;
// //                             break;
// //                         }
// //                     }
// //                     if (!can_place) continue;

// //                     // Column check
// //                     for (int x = 0; x < 9; x++) {
// //                         if (board[x * 9 + j] == num) {
// //                             can_place = false;
// //                             break;
// //                         }
// //                     }
// //                     if (!can_place) continue;

// //                     // 3x3 box check
// //                     int box_row_start = (i / 3) * 3;
// //                     int box_col_start = (j / 3) * 3;
// //                     for (int rr = box_row_start; rr < box_row_start + 3 && can_place; rr++) {
// //                         for (int cc = box_col_start; cc < box_col_start + 3; cc++) {
// //                             if (board[rr * 9 + cc] == num) {
// //                                 can_place = false;
// //                                 break;
// //                             }
// //                         }
// //                     }

// //                     if (can_place) domain_size++;
// //                     if (domain_size > 1 && domain_size >= min_domain_size) {
// //                         // If domain_size already exceeds current min_domain_size (or is >1 and min_domain_size=1)
// //                         // no need to check further for this cell
// //                         break;
// //                     }
// //                 }

// //                 // If domain_size < min_domain_size, update
// //                 if (domain_size < min_domain_size) {
// //                     min_domain_size = domain_size;
// //                     best_row = i;
// //                     best_col = j;
// //                     // If domain_size == 1, return immediately
// //                     if (domain_size == 1) {
// //                         *row = best_row;
// //                         *col = best_col;
// //                         return true;
// //                     }
// //                 }
// //             }
// //         }
// //     }

// //     // If no empty cells found, puzzle is solved
// //     if (best_row == -1 && best_col == -1) {
// //         return false; 
// //     }

// //     // Return the cell with minimal domain size found
// //     *row = best_row;
// //     *col = best_col;
// //     return true;
// // }

// // Explicit backtracking implementation for solving Sudoku
// __device__ bool solve(int *board) {
//     int stack[GRID_SIZE][2];
//     int top = -1;
//     int row, col;

//     if (!find_empty(board, &row, &col)) {
//         return true; // No empty cells, puzzle solved
//     }

//     stack[++top][0] = row;
//     stack[top][1] = col;

//     while (top >= 0) {
//         row = stack[top][0];
//         col = stack[top][1];

//         bool placed = false;
//         for (int num = board[row * 9 + col] + 1; num <= 9; num++) {
//             if (is_valid(board, row, col, num)) {
//                 board[row * 9 + col] = num;
//                 placed = true;
//                 break;
//             }
//         }

//         if (placed) {
//             if (find_empty(board, &row, &col)) {
//                 stack[++top][0] = row;
//                 stack[top][1] = col;
//             } else {
//                 return true; // Solved
//             }
//         } else {
//             board[stack[top][0] * 9 + stack[top][1]] = 0; // Reset cell
//             top--; // Backtrack
//         }
//     }

//     return false; // Unsolvable
// }

// // Host function to check if board is valid
// __device__ bool is_board_valid(int *board) {
//     // Check rows
//     for (int r = 0; r < 9; r++) {
//         int seen[10] = {0}; // track digits 1-9
//         for (int c = 0; c < 9; c++) {
//             int val = board[r*9 + c];
//             if (val != 0) {
//                 if (seen[val]) return false;
//                 seen[val] = 1;
//             }
//         }
//     }

//     // Check columns
//     for (int c = 0; c < 9; c++) {
//         int seen[10] = {0};
//         for (int r = 0; r < 9; r++) {
//             int val = board[r*9 + c];
//             if (val != 0) {
//                 if (seen[val]) return false;
//                 seen[val] = 1;
//             }
//         }
//     }

//     // Check 3x3 sub-grids
//     for (int br = 0; br < 3; br++) {
//         for (int bc = 0; bc < 3; bc++) {
//             int seen[10] = {0};
//             for (int r = br*3; r < br*3+3; r++) {
//                 for (int c = bc*3; c < bc*3+3; c++) {
//                     int val = board[r*9 + c];
//                     if (val != 0) {
//                         if (seen[val]) return false;
//                         seen[val] = 1;
//                     }
//                 }
//             }
//         }
//     }

//     // If no violations found
//     return true;
// }

// // Kernel for solving multiple Sudoku puzzles in parallel
// __global__ void solve_sudokus(int *boards, int num_boards) {
//     int idx = blockIdx.x * blockDim.x + threadIdx.x;

//     if (idx < num_boards) {
//         int *board = boards + idx * GRID_SIZE;
//         if(is_board_valid(board)) {
//             if (solve(board)) {
//                 printf("Puzzle %d solved successfully.\n", idx);
//             } else {
//                 printf("Puzzle %d is unsolvable.\n", idx);
//             }
//         }
//         else {
//             printf("Puzzle %d is unsolvable.\n", idx);
//         }
//     }
// }

// // Host function for printing a Sudoku board
// void print_board(int *board) {
//     for (int i = 0; i < 9; i++) {
//         if (i % 3 == 0 && i != 0) {
//             printf("---------------------\n");
//         }
//         for (int j = 0; j < 9; j++) {
//             if (j % 3 == 0 && j != 0) {
//                 printf("| ");
//             }
//             printf("%d ", board[i * 9 + j]);
//         }
//         printf("\n");
//     }
// }

// // Host code for managing CUDA memory and invoking the kernel
// int main() {
//     const int num_boards = 14;
//     int boards[num_boards][GRID_SIZE] = {
//         {9, 0, 0, 0, 3, 5, 0, 0, 0, 0, 0, 1, 4, 8, 0, 0, 5, 9, 3, 4, 0, 0, 0, 6, 2, 1, 0, 4, 0, 6, 5, 1, 0, 8, 3, 2, 0, 2, 0, 0, 0, 0, 6, 0, 0, 0, 0, 0, 6, 2, 8, 0, 0, 1, 0, 0, 0, 0, 0, 7, 0, 0, 4, 2, 0, 0, 9, 0, 0, 5, 8, 0, 0, 0, 0, 0, 4, 1, 9, 0, 0},
//         {0, 7, 0, 0, 0, 2, 5, 0, 9, 5, 8, 0, 3, 4, 0, 0, 0, 0, 2, 0, 1, 5, 0, 9, 0, 0, 8, 1, 0, 3, 0, 0, 0, 0, 5, 0, 9, 5, 6, 0, 3, 0, 0, 7, 1, 7, 2, 8, 0, 5, 1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 1, 4, 4, 0, 0, 0, 0, 0, 6, 0, 5, 3, 1, 5, 4, 6, 0, 0, 0, 2},
//         {8, 9, 3, 1, 4, 0, 0, 0, 0, 4, 2, 0, 3, 7, 5, 8, 1, 0, 1, 5, 0, 0, 9, 0, 2, 0, 0, 2, 0, 0, 0, 6, 7, 0, 9, 8, 0, 0, 0, 0, 3, 1, 0, 0, 0, 3, 8, 0, 5, 2, 9, 0, 7, 0, 0, 0, 1, 0, 0, 4, 0, 0, 0, 5, 0, 0, 0, 0, 0, 0, 0, 4, 0, 0, 0, 0, 8, 0, 1, 0, 0},
//         {0, 7, 0, 1, 0, 2, 0, 6, 0, 2, 0, 0, 5, 0, 0, 0, 3, 9, 0, 5, 0, 9, 0, 0, 1, 4, 0, 0, 3, 0, 4, 0, 5, 6, 8, 0, 0, 8, 5, 0, 7, 1, 0, 9, 0, 0, 0, 0, 3, 0, 0, 4, 5, 0, 7, 6, 3, 0, 0, 4, 0, 0, 0, 0, 0, 0, 7, 0, 3, 8, 1, 6, 0, 9, 0, 2, 5, 0, 3, 7, 0},
//         {7, 0, 0, 0, 0, 5, 0, 0, 0, 0, 1, 0, 3, 0, 0, 7, 2, 0, 9, 4, 0, 6, 0, 0, 1, 0, 0, 0, 5, 4, 0, 9, 1, 0, 0, 6, 0, 0, 0, 8, 7, 0, 3, 0, 0, 0, 7, 1, 5, 3, 6, 0, 4, 0, 4, 0, 6, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 5, 2, 0, 6, 9, 5, 9, 0, 4, 6, 8, 2, 0, 0},
//         {0, 7, 1, 0, 3, 0, 0, 9, 6, 0, 0, 3, 0, 6, 0, 0, 0, 5, 6, 5, 0, 7, 8, 9, 0, 0, 3, 2, 0, 8, 0, 0, 0, 0, 0, 7, 1, 0, 5, 8, 7, 0, 0, 0, 0, 0, 6, 0, 0, 0, 0, 0, 4, 0, 0, 0, 6, 0, 4, 0, 2, 3, 9, 7, 3, 2, 5, 0, 0, 6, 0, 4, 4, 1, 9, 0, 2, 0, 0, 7, 8},
//         {6, 0, 0, 3, 5, 7, 8, 9, 4, 0, 0, 0, 1, 2, 0, 6, 0, 0, 0, 0, 8, 4, 0, 0, 7, 0, 0, 0, 0, 0, 0, 4, 1, 9, 8, 6, 1, 0, 0, 9, 0, 0, 0, 7, 3, 8, 9, 0, 0, 0, 0, 4, 5, 0, 0, 0, 5, 8, 7, 0, 1, 0, 9, 7, 0, 0, 5, 1, 9, 0, 0, 8, 0, 0, 1, 6, 3, 0, 0, 0, 7},
//         {0, 1, 0, 2, 4, 3, 0, 9, 7, 0, 0, 0, 8, 0, 9, 2, 0, 0, 0, 9, 0, 7, 6, 5, 4, 1, 0, 1, 6, 2, 0, 0, 0, 9, 3, 0, 0, 0, 0, 0, 0, 6, 0, 0, 1, 9, 0, 0, 0, 0, 4, 5, 8, 6, 3, 2, 0, 4, 5, 7, 0, 0, 0, 0, 0, 9, 0, 0, 0, 0, 0, 0, 0, 5, 4, 0, 0, 0, 3, 7, 0},
//         {5, 0, 0, 0, 0, 0, 0, 7, 0, 0, 0, 0, 8, 5, 6, 3, 0, 0, 0, 0, 6, 4, 0, 0, 0, 0, 0, 0, 2, 7, 6, 4, 1, 0, 0, 0, 1, 0, 5, 0, 0, 0, 2, 4, 3, 0, 0, 0, 5, 0, 3, 0, 0, 0, 0, 0, 8, 0, 0, 5, 0, 3, 7, 0, 4, 0, 3, 0, 8, 5, 9, 1, 0, 0, 3, 9, 1, 4, 0, 0, 2},
//         {6, 0, 8, 1, 3, 0, 5, 9, 0, 0, 9, 0, 0, 5, 0, 0, 1, 0, 0, 4, 5, 8, 7, 9, 0, 3, 6, 4, 0, 0, 0, 0, 1, 7, 5, 0, 2, 0, 1, 6, 0, 5, 0, 0, 0, 5, 3, 9, 0, 2, 0, 0, 4, 0, 9, 0, 3, 0, 0, 0, 0, 0, 5, 0, 1, 0, 0, 0, 2, 9, 7, 3, 0, 0, 2, 0, 1, 3, 0, 6, 0},
//         {0, 6, 0, 0, 2, 0, 0, 0, 5, 3, 0, 0, 0, 0, 8, 6, 0, 0, 0, 9, 7, 0, 5, 0, 0, 0, 0, 0, 0, 0, 2, 9, 5, 8, 0, 1, 0, 8, 0, 0, 0, 3, 0, 9, 0, 0, 0, 3, 0, 0, 0, 4, 5, 0, 0, 2, 0, 0, 0, 1, 9, 4, 0, 7, 0, 0, 5, 0, 0, 0, 0, 8, 0, 4, 1, 0, 3, 0, 5, 0, 7},
//         {3, 8, 0, 0, 0, 0, 7, 6, 0, 0, 1, 2, 6, 0, 0, 0, 8, 4, 7, 0, 0, 0, 0, 9, 1, 0, 0, 0, 0, 0, 0, 9, 7, 0, 3, 0, 8, 0, 0, 5, 4, 0, 9, 1, 0, 0, 6, 9, 1, 8, 3, 5, 0, 7, 0, 0, 8, 0, 0, 1, 6, 0, 0, 0, 7, 0, 9, 0, 8, 0, 0, 0, 5, 9, 0, 3, 0, 4, 2, 7, 0},
//         {0, 0, 6, 1, 5, 0, 0, 0, 8, 0, 7, 3, 0, 0, 8, 5, 2, 9, 0, 0, 0, 0, 7, 0, 0, 1, 0, 0, 0, 8, 0, 0, 0, 9, 0, 0, 0, 1, 0, 0, 0, 6, 4, 0, 0, 6, 0, 0, 0, 0, 0, 2, 0, 0, 0, 8, 0, 0, 0, 0, 1, 0, 0, 7, 5, 0, 6, 1, 0, 8, 0, 0, 9, 6, 1, 0, 8, 0, 7, 4, 0},
//         {4, 0, 0, 0, 0, 0, 6, 0, 5, 0, 3, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 7, 0, 0, 0, 0, 0, 0, 2, 0, 0, 0, 0, 0, 6, 0, 0, 0, 0, 0, 8, 0, 4, 0, 0, 0, 0, 0, 0, 1, 0, 0, 0, 0, 0, 0, 0, 6, 0, 3, 0, 7, 0, 5, 0, 0, 2, 0, 0, 0, 0, 0, 0, 1, 0, 4, 0, 0, 0, 0, 0}
//     };


//     int *d_boards;
//     size_t size = num_boards * GRID_SIZE * sizeof(int);

//     // Allocate device memory
//     cudaMalloc(&d_boards, size);

//     // Copy boards to device memory
//     cudaMemcpy(d_boards, boards, size, cudaMemcpyHostToDevice);

//     // Launch the kernel
//     solve_sudokus<<<num_boards, THREADS_PER_BLOCK>>>(d_boards, num_boards);

//     // Copy results back to host
//     cudaMemcpy(boards, d_boards, size, cudaMemcpyDeviceToHost);

//     // Print the solved boards
//     for(int i = 0; i < num_boards; i++) {
//         printf("Solved Board %d:\n", i);
//         print_board(boards[i]);
//     }

//     // Free device memory
//     cudaFree(d_boards);

//     return 0;
// }

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdbool.h>

#define GRID_SIZE 81
#define THREADS_PER_BLOCK 1

// CUDA built-in popcount
__device__ int popcount9(unsigned int x) {
    return __popc(x);
}

// Place a number and update masks
__device__ void place_number(int *board, int r, int c, int val, unsigned int *rowMask, unsigned int *colMask, unsigned int *boxMask) {
    board[r*9 + c] = val;
    unsigned int bit = 1 << (val-1);
    rowMask[r] |= bit;
    colMask[c] |= bit;
    boxMask[(r/3)*3+(c/3)] |= bit;
}

// Remove a number and update masks
__device__ void remove_number(int *board, int r, int c, int val, unsigned int *rowMask, unsigned int *colMask, unsigned int *boxMask) {
    board[r*9 + c] = 0;
    unsigned int bit = 1 << (val-1);
    rowMask[r] &= ~bit;
    colMask[c] &= ~bit;
    boxMask[(r/3)*3+(c/3)] &= ~bit;
}

// Initialize masks from the current board
__device__ void init_masks(int *board, unsigned int *rowMask, unsigned int *colMask, unsigned int *boxMask) {
    for (int i = 0; i < 9; i++) {
        rowMask[i] = 0;
        colMask[i] = 0;
        boxMask[i] = 0;
    }

    for (int r = 0; r < 9; r++) {
        for (int c = 0; c < 9; c++) {
            int val = board[r*9+c];
            if (val > 0) {
                unsigned int bit = 1 << (val-1);
                rowMask[r] |= bit;
                colMask[c] |= bit;
                boxMask[(r/3)*3+(c/3)] |= bit;
            }
        }
    }
}

// find_empty using bitmask logic
__device__ bool find_empty(int *board, int *row, int *col, unsigned int *rowMask, unsigned int *colMask, unsigned int *boxMask) {
    int min_domain_size = 10;
    int best_r = -1, best_c = -1;

    for (int r = 0; r < 9; r++) {
        for (int c = 0; c < 9; c++) {
            int val = board[r*9 + c];
            if (val == 0) {
                unsigned int used = rowMask[r] | colMask[c] | boxMask[(r/3)*3+(c/3)];
                unsigned int possible = (~used) & 0x1FF;
                int domain_size = popcount9(possible);

                if (domain_size == 0) {
                    // no possibilities; treat as domain_size=10 so we never pick this cell
                    domain_size = 10;
                }

                if (domain_size < min_domain_size) {
                    min_domain_size = domain_size;
                    best_r = r;
                    best_c = c;
                    if (domain_size == 1) {
                        // perfect MRV cell
                        *row = best_r;
                        *col = best_c;
                        return true;
                    }
                }
            }
        }
    }

    if (best_r == -1 && best_c == -1) {
        // no empty cell => solved
        return false;
    }

    *row = best_r;
    *col = best_c;
    return true;
}

__device__ bool solve(int *board) {
    __shared__ unsigned int rowMask[9], colMask[9], boxMask[9];
    __shared__ int stack[GRID_SIZE][3]; // (row, col, current_val)
    __shared__ int top;

    if (threadIdx.x == 0) {
        init_masks(board, rowMask, colMask, boxMask);
        top = -1;
    }
    __syncthreads();

    int r, c;
    bool found;
    if (threadIdx.x == 0) {
        found = find_empty(board, &r, &c, rowMask, colMask, boxMask);
        if (!found) {
            // no empty cell => solved
            return true;
        }
        top++;
        stack[top][0] = r;
        stack[top][1] = c;
        stack[top][2] = board[r*9+c]; // initial val (0)
    }
    __syncthreads();

    while (true) {
        if (threadIdx.x == 0) {
            if (top < 0) {
                // no solution
                stack[0][0] = -999;
            }
        }
        __syncthreads();

        if (threadIdx.x == 0 && stack[0][0] == -999) return false;

        if (threadIdx.x == 0) {
            r = stack[top][0];
            c = stack[top][1];
            int start_val = stack[top][2];
            bool placed = false;

            // Try next possible numbers
            unsigned int used = rowMask[r] | colMask[c] | boxMask[(r/3)*3+(c/3)];
            unsigned int possible = (~used) & 0x1FF;

            // Start from start_val+1
            for (int num = start_val+1; num <= 9; num++) {
                int bit = 1 << (num-1);
                if (possible & bit) {
                    // can place num
                    place_number(board, r, c, num, rowMask, colMask, boxMask);
                    stack[top][2] = num;
                    placed = true;
                    break;
                }
            }

            if (placed) {
                bool next_found = find_empty(board, &r, &c, rowMask, colMask, boxMask);
                if (!next_found) {
                    // no empty cell => solved
                    stack[0][0] = -998;
                } else {
                    // got a cell
                    top++;
                    stack[top][0] = r;
                    stack[top][1] = c;
                    stack[top][2] = board[r*9+c];
                }
            } else {
                // no number placed, backtrack
                int val = board[r*9+c];
                if (val > 0) remove_number(board, r, c, val, rowMask, colMask, boxMask);
                top--;
            }
        }
        __syncthreads();

        if (threadIdx.x == 0 && stack[0][0] == -998) return true; // solved
        __syncthreads();
    }

    return false; // unreachable
}

__device__ bool is_board_valid(int *board) {
    // Check rows
    for (int r = 0; r < 9; r++) {
        int seen[10] = {0};
        for (int c = 0; c < 9; c++) {
            int val = board[r*9+c];
            if (val != 0) {
                if (seen[val]) return false;
                seen[val] = 1;
            }
        }
    }

    // Check cols
    for (int c = 0; c < 9; c++) {
        int seen[10] = {0};
        for (int r = 0; r < 9; r++) {
            int val = board[r*9+c];
            if (val != 0) {
                if (seen[val]) return false;
                seen[val] = 1;
            }
        }
    }

    // Check boxes
    for (int br = 0; br < 3; br++) {
        for (int bc = 0; bc < 3; bc++) {
            int seen[10] = {0};
            for (int r = br*3; r < br*3+3; r++) {
                for (int c = bc*3; c < bc*3+3; c++) {
                    int val = board[r*9+c];
                    if (val != 0) {
                        if (seen[val]) return false;
                        seen[val] = 1;
                    }
                }
            }
        }
    }

    return true;
}

__global__ void solve_sudokus(int *boards, int num_boards) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < num_boards) {
        int *board = boards + idx * GRID_SIZE;
        if(is_board_valid(board)) {
            if (solve(board)) {
                printf("Puzzle %d solved successfully.\n", idx);
            } else {
                printf("Puzzle %d is unsolvable.\n", idx);
            }
        } else {
            printf("Puzzle %d is unsolvable.\n", idx);
        }
    }
}

void print_board(int *board) {
    for (int i = 0; i < 9; i++) {
        if (i%3==0 && i!=0) printf("---------------------\n");
        for (int j = 0; j<9; j++){
            if (j%3==0 && j!=0) printf("| ");
            printf("%d ", board[i*9+j]);
        }
        printf("\n");
    }
}

int main() {
    const int num_boards = 14;
    int boards[num_boards][GRID_SIZE] = {
        {9, 0, 0, 0, 3, 5, 0, 0, 0, 0, 0, 1, 4, 8, 0, 0, 5, 9, 3, 4, 0, 0, 0, 6, 2, 1, 0, 4, 0, 6, 5, 1, 0, 8, 3, 2, 0, 2, 0, 0, 0, 0, 6, 0, 0, 0, 0, 0, 6, 2, 8, 0, 0, 1, 0, 0, 0, 0, 0, 7, 0, 0, 4, 2, 0, 0, 9, 0, 0, 5, 8, 0, 0, 0, 0, 0, 4, 1, 9, 0, 0},
        {0, 7, 0, 0, 0, 2, 5, 0, 9, 5, 8, 0, 3, 4, 0, 0, 0, 0, 2, 0, 1, 5, 0, 9, 0, 0, 8, 1, 0, 3, 0, 0, 0, 0, 5, 0, 9, 5, 6, 0, 3, 0, 0, 7, 1, 7, 2, 8, 0, 5, 1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 1, 4, 4, 0, 0, 0, 0, 0, 6, 0, 5, 3, 1, 5, 4, 6, 0, 0, 0, 2},
        {8, 9, 3, 1, 4, 0, 0, 0, 0, 4, 2, 0, 3, 7, 5, 8, 1, 0, 1, 5, 0, 0, 9, 0, 2, 0, 0, 2, 0, 0, 0, 6, 7, 0, 9, 8, 0, 0, 0, 0, 3, 1, 0, 0, 0, 3, 8, 0, 5, 2, 9, 0, 7, 0, 0, 0, 1, 0, 0, 4, 0, 0, 0, 5, 0, 0, 0, 0, 0, 0, 0, 4, 0, 0, 0, 0, 8, 0, 1, 0, 0},
        {0, 7, 0, 1, 0, 2, 0, 6, 0, 2, 0, 0, 5, 0, 0, 0, 3, 9, 0, 5, 0, 9, 0, 0, 1, 4, 0, 0, 3, 0, 4, 0, 5, 6, 8, 0, 0, 8, 5, 0, 7, 1, 0, 9, 0, 0, 0, 0, 3, 0, 0, 4, 5, 0, 7, 6, 3, 0, 0, 4, 0, 0, 0, 0, 0, 0, 7, 0, 3, 8, 1, 6, 0, 9, 0, 2, 5, 0, 3, 7, 0},
        {7, 0, 0, 0, 0, 5, 0, 0, 0, 0, 1, 0, 3, 0, 0, 7, 2, 0, 9, 4, 0, 6, 0, 0, 1, 0, 0, 0, 5, 4, 0, 9, 1, 0, 0, 6, 0, 0, 0, 8, 7, 0, 3, 0, 0, 0, 7, 1, 5, 3, 6, 0, 4, 0, 4, 0, 6, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 5, 2, 0, 6, 9, 5, 9, 0, 4, 6, 8, 2, 0, 0},
        {0, 7, 1, 0, 3, 0, 0, 9, 6, 0, 0, 3, 0, 6, 0, 0, 0, 5, 6, 5, 0, 7, 8, 9, 0, 0, 3, 2, 0, 8, 0, 0, 0, 0, 0, 7, 1, 0, 5, 8, 7, 0, 0, 0, 0, 0, 6, 0, 0, 0, 0, 0, 4, 0, 0, 0, 6, 0, 4, 0, 2, 3, 9, 7, 3, 2, 5, 0, 0, 6, 0, 4, 4, 1, 9, 0, 2, 0, 0, 7, 8},
        {6, 0, 0, 3, 5, 7, 8, 9, 4, 0, 0, 0, 1, 2, 0, 6, 0, 0, 0, 0, 8, 4, 0, 0, 7, 0, 0, 0, 0, 0, 0, 4, 1, 9, 8, 6, 1, 0, 0, 9, 0, 0, 0, 7, 3, 8, 9, 0, 0, 0, 0, 4, 5, 0, 0, 0, 5, 8, 7, 0, 1, 0, 9, 7, 0, 0, 5, 1, 9, 0, 0, 8, 0, 0, 1, 6, 3, 0, 0, 0, 7},
        {0, 1, 0, 2, 4, 3, 0, 9, 7, 0, 0, 0, 8, 0, 9, 2, 0, 0, 0, 9, 0, 7, 6, 5, 4, 1, 0, 1, 6, 2, 0, 0, 0, 9, 3, 0, 0, 0, 0, 0, 0, 6, 0, 0, 1, 9, 0, 0, 0, 0, 4, 5, 8, 6, 3, 2, 0, 4, 5, 7, 0, 0, 0, 0, 0, 9, 0, 0, 0, 0, 0, 0, 0, 5, 4, 0, 0, 0, 3, 7, 0},
        {5, 0, 0, 0, 0, 0, 0, 7, 0, 0, 0, 0, 8, 5, 6, 3, 0, 0, 0, 0, 6, 4, 0, 0, 0, 0, 0, 0, 2, 7, 6, 4, 1, 0, 0, 0, 1, 0, 5, 0, 0, 0, 2, 4, 3, 0, 0, 0, 5, 0, 3, 0, 0, 0, 0, 0, 8, 0, 0, 5, 0, 3, 7, 0, 4, 0, 3, 0, 8, 5, 9, 1, 0, 0, 3, 9, 1, 4, 0, 0, 2},
        {6, 0, 8, 1, 3, 0, 5, 9, 0, 0, 9, 0, 0, 5, 0, 0, 1, 0, 0, 4, 5, 8, 7, 9, 0, 3, 6, 4, 0, 0, 0, 0, 1, 7, 5, 0, 2, 0, 1, 6, 0, 5, 0, 0, 0, 5, 3, 9, 0, 2, 0, 0, 4, 0, 9, 0, 3, 0, 0, 0, 0, 0, 5, 0, 1, 0, 0, 0, 2, 9, 7, 3, 0, 0, 2, 0, 1, 3, 0, 6, 0},
        {0, 6, 0, 0, 2, 0, 0, 0, 5, 3, 0, 0, 0, 0, 8, 6, 0, 0, 0, 9, 7, 0, 5, 0, 0, 0, 0, 0, 0, 0, 2, 9, 5, 8, 0, 1, 0, 8, 0, 0, 0, 3, 0, 9, 0, 0, 0, 3, 0, 0, 0, 4, 5, 0, 0, 2, 0, 0, 0, 1, 9, 4, 0, 7, 0, 0, 5, 0, 0, 0, 0, 8, 0, 4, 1, 0, 3, 0, 5, 0, 7},
        {3, 8, 0, 0, 0, 0, 7, 6, 0, 0, 1, 2, 6, 0, 0, 0, 8, 4, 7, 0, 0, 0, 0, 9, 1, 0, 0, 0, 0, 0, 0, 9, 7, 0, 3, 0, 8, 0, 0, 5, 4, 0, 9, 1, 0, 0, 6, 9, 1, 8, 3, 5, 0, 7, 0, 0, 8, 0, 0, 1, 6, 0, 0, 0, 7, 0, 9, 0, 8, 0, 0, 0, 5, 9, 0, 3, 0, 4, 2, 7, 0},
        {0, 0, 6, 1, 5, 0, 0, 0, 8, 0, 7, 3, 0, 0, 8, 5, 2, 9, 0, 0, 0, 0, 7, 0, 0, 1, 0, 0, 0, 8, 0, 0, 0, 9, 0, 0, 0, 1, 0, 0, 0, 6, 4, 0, 0, 6, 0, 0, 0, 0, 0, 2, 0, 0, 0, 8, 0, 0, 0, 0, 1, 0, 0, 7, 5, 0, 6, 1, 0, 8, 0, 0, 9, 6, 1, 0, 8, 0, 7, 4, 0},
        {4, 0, 0, 0, 0, 0, 6, 0, 5, 0, 3, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 7, 0, 0, 0, 0, 0, 0, 2, 0, 0, 0, 0, 0, 6, 0, 0, 0, 0, 0, 8, 0, 4, 0, 0, 0, 0, 0, 0, 1, 0, 0, 0, 0, 0, 0, 0, 6, 0, 3, 0, 7, 0, 5, 0, 0, 2, 0, 0, 0, 0, 0, 0, 1, 0, 4, 0, 0, 0, 0, 0}
    };

    int *d_boards;
    size_t size = num_boards * GRID_SIZE * sizeof(int);

    hipMalloc(&d_boards, size);
    hipMemcpy(d_boards, boards, size, hipMemcpyHostToDevice);

    solve_sudokus<<<num_boards, THREADS_PER_BLOCK>>>(d_boards, num_boards);
    hipDeviceSynchronize();

    hipMemcpy(boards, d_boards, size, hipMemcpyDeviceToHost);

    for (int i = 0; i < num_boards; i++) {
        printf("Solved Board %d:\n", i);
        print_board(boards[i]);
        printf("\n");
    }

    hipFree(d_boards);
    return 0;
}
