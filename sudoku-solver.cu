#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdbool.h>

#define GRID_SIZE 81
#define THREADS_PER_BLOCK 1

// Helper function to check if placing a number is valid
__device__ bool is_valid(int *board, int row, int col, int num) {
    // Check the row
    for (int i = 0; i < 9; i++) {
        if (board[row * 9 + i] == num) {
            return false;
        }
    }

    // Check the column
    for (int i = 0; i < 9; i++) {
        if (board[i * 9 + col] == num) {
            return false;
        }
    }

    // Check the 3x3 sub-grid
    int box_row_start = (row / 3) * 3;
    int box_col_start = (col / 3) * 3;
    for (int i = box_row_start; i < box_row_start + 3; i++) {
        for (int j = box_col_start; j < box_col_start + 3; j++) {
            if (board[i * 9 + j] == num) {
                return false;
            }
        }
    }

    return true;
}

// Device function to find the next empty cell
__device__ bool find_empty(int *board, int *row, int *col) {
    int min_domain_size = 10; // Start with a value greater than the max possible domain size (9)
    int best_row = -1;
    int best_col = -1;

    // Iterate over all cells
    for (int i = 0; i < 9; i++) {
        for (int j = 0; j < 9; j++) {
            int val = board[i * 9 + j];
            if (val == 0) {
                // Empty cell, count possibilities
                int domain_size = 0;
                for (int num = 1; num <= 9; num++) {
                    // Check if placing 'num' is valid
                    bool can_place = true;
                    // Row check
                    for (int x = 0; x < 9; x++) {
                        if (board[i * 9 + x] == num) {
                            can_place = false;
                            break;
                        }
                    }
                    if (!can_place) continue;

                    // Column check
                    for (int x = 0; x < 9; x++) {
                        if (board[x * 9 + j] == num) {
                            can_place = false;
                            break;
                        }
                    }
                    if (!can_place) continue;

                    // 3x3 box check
                    int box_row_start = (i / 3) * 3;
                    int box_col_start = (j / 3) * 3;
                    for (int rr = box_row_start; rr < box_row_start + 3 && can_place; rr++) {
                        for (int cc = box_col_start; cc < box_col_start + 3; cc++) {
                            if (board[rr * 9 + cc] == num) {
                                can_place = false;
                                break;
                            }
                        }
                    }

                    if (can_place) domain_size++;
                    if (domain_size > 1 && domain_size >= min_domain_size) {
                        // If domain_size already exceeds current min_domain_size (or is >1 and min_domain_size=1)
                        // no need to check further for this cell
                        break;
                    }
                }

                // If domain_size < min_domain_size, update
                if (domain_size < min_domain_size) {
                    min_domain_size = domain_size;
                    best_row = i;
                    best_col = j;
                    // If domain_size == 1, return immediately
                    if (domain_size == 1) {
                        *row = best_row;
                        *col = best_col;
                        return true;
                    }
                }
            }
        }
    }

    // If no empty cells found, puzzle is solved
    if (best_row == -1 && best_col == -1) {
        return false; 
    }

    // Return the cell with minimal domain size found
    *row = best_row;
    *col = best_col;
    return true;
}

// Explicit backtracking implementation for solving Sudoku
__device__ bool solve(int *board) {
    int stack[GRID_SIZE][2];
    int top = -1;
    int row, col;

    if (!find_empty(board, &row, &col)) {
        return true; // No empty cells, puzzle solved
    }

    stack[++top][0] = row;
    stack[top][1] = col;

    while (top >= 0) {
        row = stack[top][0];
        col = stack[top][1];

        bool placed = false;
        for (int num = board[row * 9 + col] + 1; num <= 9; num++) {
            if (is_valid(board, row, col, num)) {
                board[row * 9 + col] = num;
                placed = true;
                break;
            }
        }

        if (placed) {
            if (find_empty(board, &row, &col)) {
                stack[++top][0] = row;
                stack[top][1] = col;
            } else {
                return true; // Solved
            }
        } else {
            board[stack[top][0] * 9 + stack[top][1]] = 0; // Reset cell
            top--; // Backtrack
        }
    }

    return false; // Unsolvable
}

// Host function to check if board is valid
__device__ bool is_board_valid(int *board) {
    // Check rows
    for (int r = 0; r < 9; r++) {
        int seen[10] = {0}; // track digits 1-9
        for (int c = 0; c < 9; c++) {
            int val = board[r*9 + c];
            if (val != 0) {
                if (seen[val]) return false;
                seen[val] = 1;
            }
        }
    }

    // Check columns
    for (int c = 0; c < 9; c++) {
        int seen[10] = {0};
        for (int r = 0; r < 9; r++) {
            int val = board[r*9 + c];
            if (val != 0) {
                if (seen[val]) return false;
                seen[val] = 1;
            }
        }
    }

    // Check 3x3 sub-grids
    for (int br = 0; br < 3; br++) {
        for (int bc = 0; bc < 3; bc++) {
            int seen[10] = {0};
            for (int r = br*3; r < br*3+3; r++) {
                for (int c = bc*3; c < bc*3+3; c++) {
                    int val = board[r*9 + c];
                    if (val != 0) {
                        if (seen[val]) return false;
                        seen[val] = 1;
                    }
                }
            }
        }
    }

    // If no violations found
    return true;
}

// Kernel for solving multiple Sudoku puzzles in parallel
__global__ void solve_sudokus(int *boards, int num_boards) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < num_boards) {
        int *board = boards + idx * GRID_SIZE;
        if(is_board_valid(board)) {
            if (solve(board)) {
                printf("Puzzle %d solved successfully.\n", idx);
            } else {
                printf("Puzzle %d is unsolvable.\n", idx);
            }
        }
        else {
            printf("Puzzle %d is unsolvable.\n", idx);
        }
    }
}

// Host function for printing a Sudoku board
void print_board(int *board) {
    for (int i = 0; i < 9; i++) {
        if (i % 3 == 0 && i != 0) {
            printf("---------------------\n");
        }
        for (int j = 0; j < 9; j++) {
            if (j % 3 == 0 && j != 0) {
                printf("| ");
            }
            printf("%d ", board[i * 9 + j]);
        }
        printf("\n");
    }
}

// Host code for managing CUDA memory and invoking the kernel
int main() {
    const int num_boards = 14;
    int boards[num_boards][GRID_SIZE] = {
        {9, 0, 0, 0, 3, 5, 0, 0, 0, 0, 0, 1, 4, 8, 0, 0, 5, 9, 3, 4, 0, 0, 0, 6, 2, 1, 0, 4, 0, 6, 5, 1, 0, 8, 3, 2, 0, 2, 0, 0, 0, 0, 6, 0, 0, 0, 0, 0, 6, 2, 8, 0, 0, 1, 0, 0, 0, 0, 0, 7, 0, 0, 4, 2, 0, 0, 9, 0, 0, 5, 8, 0, 0, 0, 0, 0, 4, 1, 9, 0, 0},
        {0, 7, 0, 0, 0, 2, 5, 0, 9, 5, 8, 0, 3, 4, 0, 0, 0, 0, 2, 0, 1, 5, 0, 9, 0, 0, 8, 1, 0, 3, 0, 0, 0, 0, 5, 0, 9, 5, 6, 0, 3, 0, 0, 7, 1, 7, 2, 8, 0, 5, 1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 1, 4, 4, 0, 0, 0, 0, 0, 6, 0, 5, 3, 1, 5, 4, 6, 0, 0, 0, 2},
        {8, 9, 3, 1, 4, 0, 0, 0, 0, 4, 2, 0, 3, 7, 5, 8, 1, 0, 1, 5, 0, 0, 9, 0, 2, 0, 0, 2, 0, 0, 0, 6, 7, 0, 9, 8, 0, 0, 0, 0, 3, 1, 0, 0, 0, 3, 8, 0, 5, 2, 9, 0, 7, 0, 0, 0, 1, 0, 0, 4, 0, 0, 0, 5, 0, 0, 0, 0, 0, 0, 0, 4, 0, 0, 0, 0, 8, 0, 1, 0, 0},
        {0, 7, 0, 1, 0, 2, 0, 6, 0, 2, 0, 0, 5, 0, 0, 0, 3, 9, 0, 5, 0, 9, 0, 0, 1, 4, 0, 0, 3, 0, 4, 0, 5, 6, 8, 0, 0, 8, 5, 0, 7, 1, 0, 9, 0, 0, 0, 0, 3, 0, 0, 4, 5, 0, 7, 6, 3, 0, 0, 4, 0, 0, 0, 0, 0, 0, 7, 0, 3, 8, 1, 6, 0, 9, 0, 2, 5, 0, 3, 7, 0},
        {7, 0, 0, 0, 0, 5, 0, 0, 0, 0, 1, 0, 3, 0, 0, 7, 2, 0, 9, 4, 0, 6, 0, 0, 1, 0, 0, 0, 5, 4, 0, 9, 1, 0, 0, 6, 0, 0, 0, 8, 7, 0, 3, 0, 0, 0, 7, 1, 5, 3, 6, 0, 4, 0, 4, 0, 6, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 5, 2, 0, 6, 9, 5, 9, 0, 4, 6, 8, 2, 0, 0},
        {0, 7, 1, 0, 3, 0, 0, 9, 6, 0, 0, 3, 0, 6, 0, 0, 0, 5, 6, 5, 0, 7, 8, 9, 0, 0, 3, 2, 0, 8, 0, 0, 0, 0, 0, 7, 1, 0, 5, 8, 7, 0, 0, 0, 0, 0, 6, 0, 0, 0, 0, 0, 4, 0, 0, 0, 6, 0, 4, 0, 2, 3, 9, 7, 3, 2, 5, 0, 0, 6, 0, 4, 4, 1, 9, 0, 2, 0, 0, 7, 8},
        {6, 0, 0, 3, 5, 7, 8, 9, 4, 0, 0, 0, 1, 2, 0, 6, 0, 0, 0, 0, 8, 4, 0, 0, 7, 0, 0, 0, 0, 0, 0, 4, 1, 9, 8, 6, 1, 0, 0, 9, 0, 0, 0, 7, 3, 8, 9, 0, 0, 0, 0, 4, 5, 0, 0, 0, 5, 8, 7, 0, 1, 0, 9, 7, 0, 0, 5, 1, 9, 0, 0, 8, 0, 0, 1, 6, 3, 0, 0, 0, 7},
        {0, 1, 0, 2, 4, 3, 0, 9, 7, 0, 0, 0, 8, 0, 9, 2, 0, 0, 0, 9, 0, 7, 6, 5, 4, 1, 0, 1, 6, 2, 0, 0, 0, 9, 3, 0, 0, 0, 0, 0, 0, 6, 0, 0, 1, 9, 0, 0, 0, 0, 4, 5, 8, 6, 3, 2, 0, 4, 5, 7, 0, 0, 0, 0, 0, 9, 0, 0, 0, 0, 0, 0, 0, 5, 4, 0, 0, 0, 3, 7, 0},
        {5, 0, 0, 0, 0, 0, 0, 7, 0, 0, 0, 0, 8, 5, 6, 3, 0, 0, 0, 0, 6, 4, 0, 0, 0, 0, 0, 0, 2, 7, 6, 4, 1, 0, 0, 0, 1, 0, 5, 0, 0, 0, 2, 4, 3, 0, 0, 0, 5, 0, 3, 0, 0, 0, 0, 0, 8, 0, 0, 5, 0, 3, 7, 0, 4, 0, 3, 0, 8, 5, 9, 1, 0, 0, 3, 9, 1, 4, 0, 0, 2},
        {6, 0, 8, 1, 3, 0, 5, 9, 0, 0, 9, 0, 0, 5, 0, 0, 1, 0, 0, 4, 5, 8, 7, 9, 0, 3, 6, 4, 0, 0, 0, 0, 1, 7, 5, 0, 2, 0, 1, 6, 0, 5, 0, 0, 0, 5, 3, 9, 0, 2, 0, 0, 4, 0, 9, 0, 3, 0, 0, 0, 0, 0, 5, 0, 1, 0, 0, 0, 2, 9, 7, 3, 0, 0, 2, 0, 1, 3, 0, 6, 0},
        {0, 6, 0, 0, 2, 0, 0, 0, 5, 3, 0, 0, 0, 0, 8, 6, 0, 0, 0, 9, 7, 0, 5, 0, 0, 0, 0, 0, 0, 0, 2, 9, 5, 8, 0, 1, 0, 8, 0, 0, 0, 3, 0, 9, 0, 0, 0, 3, 0, 0, 0, 4, 5, 0, 0, 2, 0, 0, 0, 1, 9, 4, 0, 7, 0, 0, 5, 0, 0, 0, 0, 8, 0, 4, 1, 0, 3, 0, 5, 0, 7},
        {3, 8, 0, 0, 0, 0, 7, 6, 0, 0, 1, 2, 6, 0, 0, 0, 8, 4, 7, 0, 0, 0, 0, 9, 1, 0, 0, 0, 0, 0, 0, 9, 7, 0, 3, 0, 8, 0, 0, 5, 4, 0, 9, 1, 0, 0, 6, 9, 1, 8, 3, 5, 0, 7, 0, 0, 8, 0, 0, 1, 6, 0, 0, 0, 7, 0, 9, 0, 8, 0, 0, 0, 5, 9, 0, 3, 0, 4, 2, 7, 0},
        {0, 0, 6, 1, 5, 0, 0, 0, 8, 0, 7, 3, 0, 0, 8, 5, 2, 9, 0, 0, 0, 0, 7, 0, 0, 1, 0, 0, 0, 8, 0, 0, 0, 9, 0, 0, 0, 1, 0, 0, 0, 6, 4, 0, 0, 6, 0, 0, 0, 0, 0, 2, 0, 0, 0, 8, 0, 0, 0, 0, 1, 0, 0, 7, 5, 0, 6, 1, 0, 8, 0, 0, 9, 6, 1, 0, 8, 0, 7, 4, 0},
        {4, 0, 0, 0, 0, 0, 6, 0, 5, 0, 3, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 7, 0, 0, 0, 0, 0, 0, 2, 0, 0, 0, 0, 0, 6, 0, 0, 0, 0, 0, 8, 0, 4, 0, 0, 0, 0, 0, 0, 1, 0, 0, 0, 0, 0, 0, 0, 6, 0, 3, 0, 7, 0, 5, 0, 0, 2, 0, 0, 0, 0, 0, 0, 1, 0, 4, 0, 0, 0, 0, 0}
    };


    int *d_boards;
    size_t size = num_boards * GRID_SIZE * sizeof(int);

    // Allocate device memory
    hipMalloc(&d_boards, size);

    // Copy boards to device memory
    hipMemcpy(d_boards, boards, size, hipMemcpyHostToDevice);

    // Launch the kernel
    solve_sudokus<<<num_boards, THREADS_PER_BLOCK>>>(d_boards, num_boards);

    // Copy results back to host
    hipMemcpy(boards, d_boards, size, hipMemcpyDeviceToHost);

    // Print the solved boards
    for(int i = 0; i < num_boards; i++) {
        printf("Solved Board %d:\n", i);
        print_board(boards[i]);
    }

    // Free device memory
    hipFree(d_boards);

    return 0;
}