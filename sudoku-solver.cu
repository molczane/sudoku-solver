#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdbool.h>

#define GRID_SIZE 81
#define THREADS_PER_BLOCK 1

// Helper function to check if placing a number is valid
__device__ bool is_valid(int *board, int row, int col, int num) {
    // Check the row
    for (int i = 0; i < 9; i++) {
        if (board[row * 9 + i] == num) {
            return false;
        }
    }

    // Check the column
    for (int i = 0; i < 9; i++) {
        if (board[i * 9 + col] == num) {
            return false;
        }
    }

    // Check the 3x3 sub-grid
    int box_row_start = (row / 3) * 3;
    int box_col_start = (col / 3) * 3;
    for (int i = box_row_start; i < box_row_start + 3; i++) {
        for (int j = box_col_start; j < box_col_start + 3; j++) {
            if (board[i * 9 + j] == num) {
                return false;
            }
        }
    }

    return true;
}

// Device function to find the next empty cell
__device__ bool find_empty(int *board, int *row, int *col) {
    for (int i = 0; i < 9; i++) {
        for (int j = 0; j < 9; j++) {
            if (board[i * 9 + j] == 0) {
                *row = i;
                *col = j;
                return true;
            }
        }
    }
    return false;
}

// Explicit backtracking implementation for solving Sudoku
__device__ bool solve(int *board) {
    int possible[GRID_SIZE]; // Bitmask for possible values of each cell
    int stack[GRID_SIZE][3];      // Stack for backtracking: [cell index, bitmask, last value tried]
    int top = -1;

    // Initialize possible values for each cell
    for (int i = 0; i < GRID_SIZE; i++) {
        if (board[i] == 0) {
            int row = i / 9;
            int col = i % 9;
            int subgrid = (row / 3) * 3 + (col / 3);

            // Start with all numbers (1–9) as possible
            possible[i] = 0x1FF;

            // Remove numbers already present in the row, column, or subgrid
            for (int j = 0; j < 9; j++) {
                int val_row = board[row * 9 + j];
                int val_col = board[j * 9 + col];
                int val_subgrid = board[(subgrid / 3) * 27 + (subgrid % 3) * 3 + (j / 3) * 9 + (j % 3)];
                if (val_row > 0) possible[i] &= ~(1 << (val_row - 1));
                if (val_col > 0) possible[i] &= ~(1 << (val_col - 1));
                if (val_subgrid > 0) possible[i] &= ~(1 << (val_subgrid - 1));
            }
        } else {
            possible[i] = 0; // Filled cells have no possibilities
        }
    }

    // Backtracking loop
    while (true) {
        // Find the cell with the least number of possible values (MRV heuristic)
        int min_index = -1;
        int min_count = 10; // More than the maximum possible (9)

        for (int i = 0; i < GRID_SIZE; i++) {
            if (board[i] == 0) {
                int count = __popc(possible[i]); // Count set bits in the bitmask
                if (count > 0 && count < min_count) {
                    min_count = count;
                    min_index = i;
                }
            }
        }

        // If no cell is left, the board is solved
        if (min_index == -1) return true;

        // Get the possible values for the selected cell
        int mask = possible[min_index];
        int row = min_index / 9;
        int col = min_index % 9;
        int subgrid = (row / 3) * 3 + (col / 3);

        // Push the cell onto the stack for backtracking
        stack[++top][0] = min_index;
        stack[top][1] = mask;
        stack[top][2] = 0; // Start with the first possible value

        while (top >= 0) {
            int cell = stack[top][0];
            mask = stack[top][1];
            int last_value = stack[top][2];

            // Find the next possible value for the cell
            int next_value = -1;
            for (int num = last_value + 1; num <= 9; num++) {
                if (mask & (1 << (num - 1))) {
                    next_value = num;
                    break;
                }
            }

            if (next_value == -1) {
                // No more possible values, backtrack
                board[cell] = 0;
                possible[cell] = stack[top--][1]; // Restore possibilities
            } else {
                // Place the value in the cell
                board[cell] = next_value;
                stack[top][2] = next_value;

                // Update constraints dynamically
                int bit = 1 << (next_value - 1);
                for (int j = 0; j < 9; j++) {
                    int row_cell = row * 9 + j;
                    int col_cell = j * 9 + col;
                    int subgrid_cell = (subgrid / 3) * 27 + (subgrid % 3) * 3 + (j / 3) * 9 + (j % 3);
                    possible[row_cell] &= ~bit;
                    possible[col_cell] &= ~bit;
                    possible[subgrid_cell] &= ~bit;
                }

                // Move to the next empty cell
                break;
            }
        }

        // If the stack is empty and no solution is found, the puzzle is unsolvable
        if (top < 0) return false;
    }
}

// Kernel for solving multiple Sudoku puzzles in parallel
__global__ void solve_sudokus(int *boards, int num_boards) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < num_boards) {
        int *board = boards + idx * GRID_SIZE;
        if (solve(board)) {
            printf("Puzzle %d solved successfully.\n", idx);
        } else {
            printf("Puzzle %d is unsolvable.\n", idx);
        }
    }
}

// Host function for printing a Sudoku board
void print_board(int *board) {
    for (int i = 0; i < 9; i++) {
        for (int j = 0; j < 9; j++) {
            printf("%d ", board[i * 9 + j]);
        }
        printf("\n");
    }
}

// Host code for managing CUDA memory and invoking the kernel
int main() {
    const int num_boards = 13;
int boards[num_boards][GRID_SIZE] = {
    {9, 0, 0, 0, 1, 5, 0, 0, 0, 0, 0, 1, 4, 8, 0, 0, 5, 9, 3, 4, 0, 0, 0, 6, 2, 1, 0, 4, 0, 6, 5, 1, 0, 8, 3, 2, 0, 2, 0, 0, 0, 0, 6, 0, 0, 0, 0, 0, 6, 2, 8, 0, 0, 1, 0, 0, 0, 0, 0, 7, 0, 0, 4, 2, 0, 0, 9, 0, 0, 5, 8, 0, 0, 0, 0, 0, 4, 1, 9, 0, 0},
    {0, 7, 0, 0, 0, 2, 5, 0, 9, 5, 8, 0, 3, 4, 0, 0, 0, 0, 2, 0, 1, 5, 0, 9, 0, 0, 8, 1, 0, 3, 0, 0, 0, 0, 5, 0, 9, 5, 6, 0, 3, 0, 0, 7, 1, 7, 2, 8, 0, 5, 1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 1, 4, 4, 0, 0, 0, 0, 0, 6, 0, 5, 3, 1, 5, 4, 6, 0, 0, 0, 2},
    {8, 9, 3, 1, 4, 0, 0, 0, 0, 4, 2, 0, 3, 7, 5, 8, 1, 0, 1, 5, 0, 0, 9, 0, 2, 0, 0, 2, 0, 0, 0, 6, 7, 0, 9, 8, 0, 0, 0, 0, 3, 1, 0, 0, 0, 3, 8, 0, 5, 2, 9, 0, 7, 0, 0, 0, 1, 0, 0, 4, 0, 0, 0, 5, 0, 0, 0, 0, 0, 0, 0, 4, 0, 0, 0, 0, 8, 0, 1, 0, 0},
    {0, 7, 0, 1, 0, 2, 0, 6, 0, 2, 0, 0, 5, 0, 0, 0, 3, 9, 0, 5, 0, 9, 0, 0, 1, 4, 0, 0, 3, 0, 4, 0, 5, 6, 8, 0, 0, 8, 5, 0, 7, 1, 0, 9, 0, 0, 0, 0, 3, 0, 0, 4, 5, 0, 7, 6, 3, 0, 0, 4, 0, 0, 0, 0, 0, 0, 7, 0, 3, 8, 1, 6, 0, 9, 0, 2, 5, 0, 3, 7, 0},
    {7, 0, 0, 0, 0, 5, 0, 0, 0, 0, 1, 0, 3, 0, 0, 7, 2, 0, 9, 4, 0, 6, 0, 0, 1, 0, 0, 0, 5, 4, 0, 9, 1, 0, 0, 6, 0, 0, 0, 8, 7, 0, 3, 0, 0, 0, 7, 1, 5, 3, 6, 0, 4, 0, 4, 0, 6, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 5, 2, 0, 6, 9, 5, 9, 0, 4, 6, 8, 2, 0, 0},
    {0, 7, 1, 0, 3, 0, 0, 9, 6, 0, 0, 3, 0, 6, 0, 0, 0, 5, 6, 5, 0, 7, 8, 9, 0, 0, 3, 2, 0, 8, 0, 0, 0, 0, 0, 7, 1, 0, 5, 8, 7, 0, 0, 0, 0, 0, 6, 0, 0, 0, 0, 0, 4, 0, 0, 0, 6, 0, 4, 0, 2, 3, 9, 7, 3, 2, 5, 0, 0, 6, 0, 4, 4, 1, 9, 0, 2, 0, 0, 7, 8},
    {6, 0, 0, 3, 5, 7, 8, 9, 4, 0, 0, 0, 1, 2, 0, 6, 0, 0, 0, 0, 8, 4, 0, 0, 7, 0, 0, 0, 0, 0, 0, 4, 1, 9, 8, 6, 1, 0, 0, 9, 0, 0, 0, 7, 3, 8, 9, 0, 0, 0, 0, 4, 5, 0, 0, 0, 5, 8, 7, 0, 1, 0, 9, 7, 0, 0, 5, 1, 9, 0, 0, 8, 0, 0, 1, 6, 3, 0, 0, 0, 7},
    {0, 1, 0, 2, 4, 3, 0, 9, 7, 0, 0, 0, 8, 0, 9, 2, 0, 0, 0, 9, 0, 7, 6, 5, 4, 1, 0, 1, 6, 2, 0, 0, 0, 9, 3, 0, 0, 0, 0, 0, 0, 6, 0, 0, 1, 9, 0, 0, 0, 0, 4, 5, 8, 6, 3, 2, 0, 4, 5, 7, 0, 0, 0, 0, 0, 9, 0, 0, 0, 0, 0, 0, 0, 5, 4, 0, 0, 0, 3, 7, 0},
    {5, 0, 0, 0, 0, 0, 0, 7, 0, 0, 0, 0, 8, 5, 6, 3, 0, 0, 0, 0, 6, 4, 0, 0, 0, 0, 0, 0, 2, 7, 6, 4, 1, 0, 0, 0, 1, 0, 5, 0, 0, 0, 2, 4, 3, 0, 0, 0, 5, 0, 3, 0, 0, 0, 0, 0, 8, 0, 0, 5, 0, 3, 7, 0, 4, 0, 3, 0, 8, 5, 9, 1, 0, 0, 3, 9, 1, 4, 0, 0, 2},
    {6, 0, 8, 1, 3, 0, 5, 9, 0, 0, 9, 0, 0, 5, 0, 0, 1, 0, 0, 4, 5, 8, 7, 9, 0, 3, 6, 4, 0, 0, 0, 0, 1, 7, 5, 0, 2, 0, 1, 6, 0, 5, 0, 0, 0, 5, 3, 9, 0, 2, 0, 0, 4, 0, 9, 0, 3, 0, 0, 0, 0, 0, 5, 0, 1, 0, 0, 0, 2, 9, 7, 3, 0, 0, 2, 0, 1, 3, 0, 6, 0},
    {0, 6, 0, 0, 2, 0, 0, 0, 5, 3, 0, 0, 0, 0, 8, 6, 0, 0, 0, 9, 7, 0, 5, 0, 0, 0, 0, 0, 0, 0, 2, 9, 5, 8, 0, 1, 0, 8, 0, 0, 0, 3, 0, 9, 0, 0, 0, 3, 0, 0, 0, 4, 5, 0, 0, 2, 0, 0, 0, 1, 9, 4, 0, 7, 0, 0, 5, 0, 0, 0, 0, 8, 0, 4, 1, 0, 3, 0, 5, 0, 7},
    {3, 8, 0, 0, 0, 0, 7, 6, 0, 0, 1, 2, 6, 0, 0, 0, 8, 4, 7, 0, 0, 0, 0, 9, 1, 0, 0, 0, 0, 0, 0, 9, 7, 0, 3, 0, 8, 0, 0, 5, 4, 0, 9, 1, 0, 0, 6, 9, 1, 8, 3, 5, 0, 7, 0, 0, 8, 0, 0, 1, 6, 0, 0, 0, 7, 0, 9, 0, 8, 0, 0, 0, 5, 9, 0, 3, 0, 4, 2, 7, 0},
    {0, 0, 6, 1, 5, 0, 0, 0, 8, 0, 7, 3, 0, 0, 8, 5, 2, 9, 0, 0, 0, 0, 7, 0, 0, 1, 0, 0, 0, 8, 0, 0, 0, 9, 0, 0, 0, 1, 0, 0, 0, 6, 4, 0, 0, 6, 0, 0, 0, 0, 0, 2, 0, 0, 0, 8, 0, 0, 0, 0, 1, 0, 0, 7, 5, 0, 6, 1, 0, 8, 0, 0, 9, 6, 1, 0, 8, 0, 7, 4, 0}
};


    int *d_boards;
    size_t size = num_boards * GRID_SIZE * sizeof(int);

    // Allocate device memory
    hipMalloc(&d_boards, size);

    // Copy boards to device memory
    hipMemcpy(d_boards, boards, size, hipMemcpyHostToDevice);

    // Launch the kernel
    solve_sudokus<<<num_boards, THREADS_PER_BLOCK>>>(d_boards, num_boards);

    // Copy results back to host
    hipMemcpy(boards, d_boards, size, hipMemcpyDeviceToHost);

    // Print the solved boards
    for(int i = 0; i < num_boards; i++) {
        printf("Solved Board %d:\n", i);
        print_board(boards[i]);
    }

    // Free device memory
    hipFree(d_boards);

    return 0;
}
